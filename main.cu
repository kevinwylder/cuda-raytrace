#include "hip/hip_runtime.h"
#include "util.c"
#include "math.h"
#include "image.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/stat.h>
#include <unistd.h>

void allocateImage(Image *img, size_t width, size_t height)
{
    img->width = width;
    img->height = height;
    for (int channel = 0; channel < 3; channel++)
    {
        cudaCheck(hipMallocPitch((void **)&img->data.channel[channel], &img->data.pitch[channel], width, height));
    }

    float phi = 1.6180339887;
    float phi_inv = 1.0 / phi;

    // https://upload.wikimedia.org/wikipedia/commons/a/a4/Dodecahedron_vertices.png
    img->points[0] = {-1, 1, 1};
    img->points[1] = {1, 1, 1};
    img->points[2] = {-1, -1, 1};
    img->points[3] = {1, -1, 1};
    img->points[4] = {-1, 1, -1};
    img->points[5] = {1, 1, -1};
    img->points[6] = {-1, -1, -1};
    img->points[7] = {1, -1, -1};
    img->points[8] = {-phi, 0, phi_inv};
    img->points[9] = {phi, 0, phi_inv};
    img->points[10] = {-phi, 0, -phi_inv};
    img->points[11] = {phi, 0, -phi_inv};
    img->points[12] = {-phi_inv, phi, 0};
    img->points[13] = {phi_inv, phi, 0};
    img->points[14] = {-phi_inv, -phi, 0};
    img->points[15] = {phi_inv, -phi, 0};
    img->points[16] = {0, phi_inv, phi};
    img->points[17] = {0, -phi_inv, phi};
    img->points[18] = {0, phi_inv, -phi};
    img->points[19] = {0, -phi_inv, -phi};

    img->faces[0] = {17, 16, 1, 9, 3};
    img->faces[1] = {1, 16, 0, 12, 13};
    img->faces[2] = {1, 13, 5, 11, 9};
    img->faces[3] = {9, 11, 7, 15, 3};
    img->faces[4] = {17, 3, 15, 14, 2};
    img->faces[5] = {2, 8, 0, 16, 17};
    img->faces[6] = {14, 6, 10, 8, 2};
    img->faces[7] = {4, 12, 0, 8, 10};
    img->faces[8] = {13, 12, 4, 18, 5};
    img->faces[9] = {5, 18, 19, 7, 11};
    img->faces[10] = {6, 14, 15, 7, 19};
    img->faces[11] = {10, 6, 19, 18, 4};
}

typedef struct Ray
{
    Point location;
    Point direction;
} Ray;

__device__ bool collides(Point *a, Point *b, Point *c, Ray *ray)
{
    // https://stackoverflow.com/questions/42740765/intersection-between-line-and-triangle-in-3d
    Point e1, e2, n, ao, dao;
    sub(&e1, b, a);
    sub(&e2, c, a);
    cross(&n, &e1, &e2);
    sub(&ao, &ray->location, a);
    cross(&dao, &ao, &ray->direction);
    float det = -dot(&ray->direction, &n);
    float invdet = 1.0 / det;
    float u = dot(&e2, &dao) * invdet;
    float v = -dot(&e1, &dao) * invdet;
    float t = dot(&ao, &n) * invdet;
    return (det >= 1e-6 && t >= 0.0 && u >= 0.0 && v >= 0.0 && (u + v) <= 1.0);
}

__global__ void renderPixel(Image *img)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    size_t idx = col * img->width + row;
    float x = (((float)col * 2) / img->width) - 1;
    float y = (((float)row * 2) / img->height) - 1;

    Ray ray = {
        location : {
            x : x * 2.0f,
            y : y * 2.0f,
            z : 10.,
        },
        direction : {
            x : 0,
            y : 0,
            z : -1.,
        }
    };

    img->data.channel[0][idx] = 0;
    img->data.channel[1][idx] = 0;
    img->data.channel[2][idx] = 0;

    for (size_t i = 0; i < 12; i++)
    {
        // check if we collide with a face
        Pentagon face = img->faces[i];
        bool face1 = collides(&img->points[face.a], &img->points[face.b], &img->points[face.c], &ray); // || collides(&img->points[face.a], &img->points[face.c], &img->points[face.b], &ray);
        bool face2 = collides(&img->points[face.a], &img->points[face.c], &img->points[face.d], &ray); // || collides(&img->points[face.a], &img->points[face.d], &img->points[face.c], &ray);
        bool face3 = collides(&img->points[face.a], &img->points[face.d], &img->points[face.e], &ray); // || collides(&img->points[face.a], &img->points[face.e], &img->points[face.d], &ray);
        if (face1)
        {
            img->data.channel[0][idx] = 255;
        }
        if (face2)
        {
            img->data.channel[1][idx] = 255;
        }
        if (face3)
        {
            img->data.channel[2][idx] = 255;
        }
    }
}

void renderImage(Image *img)
{
    Image *deviceImage;
    cudaCheck(hipMalloc(&deviceImage, sizeof(Image)));
    cudaCheck(hipMemcpy(deviceImage, img, sizeof(Image), hipMemcpyHostToDevice));

    dim3 threadsPerBlock(32, 32);
    dim3 blocks(img->width / threadsPerBlock.x, img->height / threadsPerBlock.y);
    renderPixel<<<blocks, threadsPerBlock>>>(deviceImage);

    cudaCheck(hipDeviceSynchronize());
    cudaCheck(hipFree(deviceImage));
}

int main()
{
    Image img{
        camera : {0., 0., -2.},
    };
    allocateImage(&img, 4096, 4096);
    debugFace(&img.points[0], &img.faces[8]);
    renderImage(&img);
    writeImageJPEG(&img, "out.jpg");
    return 0;
}
